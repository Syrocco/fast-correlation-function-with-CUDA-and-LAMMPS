#include "hip/hip_runtime.h"


#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void hello_cuda() {
        printf("hello from GPU\n");
}

int main() {
        printf("hello from CPU\n");
        hello_cuda <<<1, 1>>> ();
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );

        gpuErrchk(hipDeviceReset());
        printf("bye bye from CPU\n");
        return 0;
}
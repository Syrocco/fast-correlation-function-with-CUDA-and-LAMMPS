#include "hip/hip_runtime.h"
extern "C" { 
#include "parser.h"
}
#include <stdlib.h>
#include <math.h>

FILE* fileout;

int a, b, c;
void pars(char* data){
    char input[100];
    char *token;
    strcpy(input, data);

    // Remove the newline character from the input
    input[strcspn(input, "\n")] = '\0';

    // Use strtok to split the string
    token = strtok(input, ":");

    if (token != NULL) {
        a = atoi(token);
        token = strtok(NULL, ":");
        if (token != NULL) {
            b = atoi(token);
            token = strtok(NULL, ":");
            if (token != NULL) {
                c = atoi(token);
            }
            else {
                printf("Invalid input: Not enough values.\n");
            }
        } 
        else {
            printf("Invalid input: Not enough values.\n");
        }
    } 
    else {
        printf("Invalid input: Not enough values.\n");
    }
}

__global__ void compute_structf_kernel(float* xpos, float* ypos, float* vx, float* vy, int nscat, float* qx, int nqx, float* qy, int nqy, float* structf)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < nqx && j < nqy)
    {   
        
        printf("%d / %d \r", j, nqx);
        float im = 0;
        float re = 0;
        float q = sqrt(qx[i]*qx[i] + qy[j]*qy[j]);
        float qxu = qx[i]/q;
        float qyu = qy[j]/q;
        for (int k = 0; k < nscat; ++k)
        {
            float qr = qx[i]*xpos[k] + qy[j]*ypos[k];
            //float q = sqrt(qx[i]*qx[i] + qy[j]*qy[j]);
            //float v = qx[i]/q*vx[k] + qy[j]/q*vy[k]; 
            float T = 0.5*(vx[k]*vx[k] + vy[k]*vy[k]);
            //float v = sqrt(vx[k]*vx[k] + vy[k]*vy[k]);
            re += T*cos(qr);
            im += T*sin(qr);
        }

        structf[i * nqy + j] = (re * re + im * im) / nscat;
    }
}

void compute(Dump* dump, float qmax) {
    char hboxx;
    float L = get_boxx(&hboxx, 1, dump);
    float xx = 2*M_PI/L;
    int nq = 2*qmax/xx + 1;
    int N = get_natoms(dump);

    float* x = (float*)calloc(N, sizeof(float));
    float* y = (float*)calloc(N, sizeof(float));
    float* vx = (float*)calloc(N, sizeof(float));
    float* vy = (float*)calloc(N, sizeof(float));
    float* q = (float*)calloc(nq, sizeof(float));
    float* S = (float*)calloc(nq*nq, sizeof(float));
    for (int i = 0; i < nq; ++i){
		q[i] = xx * ( i - (nq - 1) / 2);
    }
    

    if (a < 0){
        a = 0;
    }
    if (b < 1){
        b = dump->nframes;
    }
    if (c < 1){
        c = 1;
    }
    fprintf(fileout, "%d\n", (int)((b - a) / c));
    for (int i = 0; i < nq; ++i)
        fprintf(fileout, "%g ", q[i]);
    fprintf(fileout, "\n");

    for (int j = 0; j < nq; ++j)
        fprintf(fileout, "%g ", q[j]);
    fprintf(fileout, "\n");

    float* device_x;
    float* device_y;
    float* device_vx;
    float* device_vy;
    float* device_q;
    float* device_S;
    // Allocate memory on the GPU
    hipMalloc((void**)&device_x, N * sizeof(float));
    hipMalloc((void**)&device_y, N * sizeof(float));
    hipMalloc((void**)&device_vx, N * sizeof(float));
    hipMalloc((void**)&device_vy, N * sizeof(float));
    hipMalloc((void**)&device_q, nq * sizeof(float));
    hipMalloc((void**)&device_S, nq*nq * sizeof(float));
    hipMemcpy(device_q, q, nq * sizeof(float), hipMemcpyHostToDevice);
    

    // Define the number of threads per block and the number of blocks
    dim3 blockDim(32, 32); // Adjust block dimensions as needed
    dim3 gridDim((nq + blockDim.x - 1) / blockDim.x, (nq + blockDim.y - 1) / blockDim.y);
    
    
    for (int i = a; i < b; i = i + c){
        
        jump_to_frame(i, dump);

        get_floatatomprop("x", x, N, dump);
        get_floatatomprop("y", y, N, dump);
        get_floatatomprop("vx", vx, N, dump);
        get_floatatomprop("vy", vy, N, dump);

        

        

        // Copy data from host to device
        hipMemset(device_S, 0, nq * nq * sizeof(float));
        hipMemcpy(device_x, x, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(device_y, y, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(device_vx, vx, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(device_vy, vy, N * sizeof(float), hipMemcpyHostToDevice);


        compute_structf_kernel<<<gridDim, blockDim>>>(device_x, device_y, device_vx, device_vy, N, device_q, nq, device_q, nq, device_S);

        // Copy the result back to the host
        hipMemcpy(S, device_S, nq * nq * sizeof(float), hipMemcpyDeviceToHost);

        

        /* Write the structure factor as a nqx * nqy matrix. */
        for (int i = 0; i < nq; ++i)
        {
            for (int j = 0; j < nq; ++j)
                fprintf(fileout, "%g ", S[i*nq + j]);

            fprintf(fileout, "\n");
    	}
    }
    fclose(fileout);
}


int main(int argc, char** argv){
    float qmax = atof(argv[1]);
    char* path_in = argv[2];
    char* path_out = argv[3];
    pars(argv[4]);
    fileout = fopen(path_out, "w");
    Dump* dump = dump_open(path_in, 'r');
    compute(dump, qmax);
} 

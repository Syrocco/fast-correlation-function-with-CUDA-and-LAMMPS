#include "hip/hip_runtime.h"

extern "C" { 
#include "parser.h"
}
#include <stdlib.h>
#include <math.h>

typedef struct {   
    float r;
    float g;
} corr;

int a, b, c;
void pars(char* data){
    char input[100];
    char *token;
    strcpy(input, data);

    
    input[strcspn(input, "\n")] = '\0';

  
    token = strtok(input, ":");

    if (token != NULL) {
        a = atoi(token);
        token = strtok(NULL, ":");
        if (token != NULL) {
            b = atoi(token);
            token = strtok(NULL, ":");
            if (token != NULL) {
                c = atoi(token);
            }
            else {
                printf("Invalid input: Not enough values.\n");
            }
        } 
        else {
            printf("Invalid input: Not enough values.\n");
        }
    } 
    else {
        printf("Invalid input: Not enough values.\n");
    }
}

#define floatingType double


__global__ void histogramKernel(float* x, float* y, floatingType* count, int N, float halfLx, float halfLy, int n) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    printf("%d\n", tid);
    float qx = 3.2329;
    float qy = 0.4573;
    if (tid < N) {
        //printf("%d / %d \r", tid, N);
        float xi = x[tid];
        float yi = y[tid];

        for (int j = 0; j < tid; j++) {
            float dx = xi - x[j];
            float dy = yi - y[j];
            if (dx >= halfLx)
                dx -= 2*halfLx;
            else if (dx < -halfLx)
                dx += 2*halfLx;
            if (dy >= halfLy)
                dy -= 2*halfLy;
            else if (dy < -halfLy)
                dy += 2*halfLy;
            float r = sqrt(dx * dx + dy * dy);

            if (r < min(halfLx, halfLy)) {
                int idx = (int)(n * (r / min(halfLx, halfLy)));
                atomicAdd(&count[idx], (floatingType)2*cos(qx*dx + qy*dy));
            }
        }
    }
}

corr* correlation(Dump* dump, int n) {
    char hboxx;
    char hboxy;
    float Lx = get_boxx(&hboxx, 1, dump);
    float halfLx = 0.5 * Lx;
    float Ly = get_boxy(&hboxy, 1, dump);
    float halfLy = 0.5 * Ly;
    int N = get_natoms(dump);

    float* x = (float*)calloc(N, sizeof(float));
    float* y = (float*)calloc(N, sizeof(float));
    corr* g6 = (corr*)calloc(n, sizeof(corr));

    int loopN = 0;
     if (a < 0){
        a = 0;
    }
    if (b < 1){
        b = dump->nframes;
    }
    if (c < 1){
        c = 1;
    }

    float* device_x;
    float* device_y;
    floatingType* device_count;
    hipMalloc((void**)&device_x, N * sizeof(float));
    hipMalloc((void**)&device_y, N * sizeof(float));
    hipMalloc((void**)&device_count, n * sizeof(floatingType));
    floatingType* count = (floatingType*)calloc(n, sizeof(floatingType));
    for (int frame = a; frame < b; frame += c) {
        //printf("%d/%d\r", a, b);
        fflush(stdout);
        loopN++;
        jump_to_frame(frame, dump);

        get_floatatomprop("x", x, N, dump);
        get_floatatomprop("y", y, N, dump);

        

        // Copy data from host to device
        hipMemcpy(device_x, x, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemcpy(device_y, y, N * sizeof(float), hipMemcpyHostToDevice);
        hipMemset(device_count, 0, n * sizeof(floatingType));

        // Define the number of threads per block and the number of blocks
        int threadsPerBlock = 256;
        int numBlocks = (N + threadsPerBlock - 1)/threadsPerBlock;

        
        histogramKernel<<<numBlocks, threadsPerBlock>>>(device_x, device_y, device_count, N, halfLx, halfLy, n);

        hipMemcpy(count, device_count, n * sizeof(floatingType), hipMemcpyDeviceToHost);

        for (int i = 0; i < n; i++){
            g6[i].g += count[i];
            count[i] = 0;
        }


    }
    // Clean up device memory
    hipFree(device_x);
    hipFree(device_y);
    hipFree(device_count);
    free(count);
    for (int i = 0; i < n; i++){
        g6[i].r = (i + 0.5)*(min(halfLx, halfLy)/n);
        g6[i].g /= loopN;
        g6[i].g = Lx*Ly*g6[i].g/(2*M_PI*N*(N - 1)*g6[i].r*(g6[1].r - g6[0].r));
        printf("%lf ", g6[i].g);
    }


    return g6;
}

void print(corr* g, int n){

}

int main(int argc, char** argv){
    float factor = atof(argv[1]);
    char* path_in = argv[2];
    char* path_out = argv[3];
    pars(argv[4]);
    size_t length = strlen(path_in);
    Dump* dump = dump_open(path_in, 'r');
    double sig = 2;
    if ('L' == path_in[length - 1]){
        sig = 0.0025;
    }
    char hboxx;
    int n = get_boxx(&hboxx, 1, dump)/(sig*2*factor);
    corr* g = correlation(dump, n);
    FILE* file;

    file = fopen (path_out, "w+");
    for (int i = 0; i < n; i++)
        fprintf(file, "%f %f\n", g[i].r, g[i].g);
    fclose(file);

}





